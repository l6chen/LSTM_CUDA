#include "hip/hip_runtime.h"
/**
*	Author: Lingfeng Chen, Pengkun Li
*   PID: A53270085, A53270024
*	This file defines the data structure for grid
*/
#include "basicLayer.h"
#define BLOCK_SIZE 32

namespace basicLayer {

	__global__ void randInitGPU(int nx, int ny, hiprandState* rndstates) {
		unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
		unsigned int iy = blockIdx.y;
		unsigned int idx = iy * nx + ix;
		if (ix < nx && iy < ny) {
			hiprand_init(clock() + idx, idx, 0, &rndstates[idx]);
		}
			
	}

	__global__ void weightbiasTruncInitGPU(int hid, int emb, float* d_Wh, float* d_Wx,
		float* d_b, hiprandState* rndstates) { // truncated uniform
		unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
		unsigned int iy = blockIdx.y;
		unsigned int idh = iy * (hid + emb) + ix;

		//for Wh
		if (ix < hid && iy < hid) {
			d_Wh[iy * hid + ix] = hiprand_uniform(&rndstates[idh]) * 2.0E-4 - 1.0E-4;
		}//for Wx
		else if (ix < hid + emb && iy < hid) {
			d_Wx[iy * emb + ix - hid] = hiprand_uniform(&rndstates[idh]) * 2.0E-4 - 1.0E-4;
		}//for bin
		else if (ix < hid && iy == hid) {
			d_b[ix] = hiprand_uniform(&rndstates[idh]) * 2.0E-4 - 1.0E-4;
		}
		        
	}

	__global__ void weightbiasGradInitGPU(int hid, int emb, float* d_Wh, float* d_Wx,
		float* d_b) { 
		unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
		unsigned int iy = blockIdx.y;
		unsigned int idh = iy * (hid + emb) + ix;

		//for Wh
		if (ix < hid && iy < hid) {
			d_Wh[iy * hid + ix] = 0.0f;
		}//for Wx
		else if (ix < hid + emb && iy < hid) {
			d_Wx[iy * emb + ix - hid] = 0.0f;
		}//for bin
		else if (ix < hid && iy == hid) {
			d_b[ix] = 0.0f;
		}

	}
	__global__ void concatVecGPU(const int nx, const int ny, float* d_out, 
		float* d_A, float* d_B){
		unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx < nx) {
			d_out[idx] = d_A[idx];
		}
		else if (idx < nx + ny) {
			d_out[idx] = d_B[idx - nx];
		}
	
	}

	__global__ void denseweightbiasTruncInitGPU(int nx, int ny, float* d_W,
		float* d_b, hiprandState* rndstates, int layerId) {// truncated uniform
		unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
		unsigned int iy = blockIdx.y;
		unsigned int idx = iy * nx + ix;

		if (layerId == 1) {//for dense
			//for W
			if (ix < nx - 1 && iy < ny) {
				d_W[iy * (nx - 1) + ix] = hiprand_uniform(&rndstates[idx]) * 2.0E-4 - 1.0E-4;
			}//for b
			else if (ix == nx - 1 && iy < ny) {
				d_b[iy] = hiprand_uniform(&rndstates[idx]) * 2.0E-4 - 1.0E-4;
			}
		}
		else {//for embed
			if (ix < nx && iy < ny)
				d_W[idx] = hiprand_uniform(&rndstates[idx]) * 2.0E-4 - 1.0E-4;
		}
	}

	__global__ void denseweightbiasGradInitGPU(int nx, int ny, float* d_W,
		float* d_b, hiprandState* rndstates, int layerId) {
		unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
		unsigned int iy = blockIdx.y;
		unsigned int idx = iy * nx + ix;

		if (layerId == 1) {//for dense
		//for W
			if (ix < nx - 1 && iy < ny) {
				d_W[iy * (nx - 1) + ix] = 0.0f;
			}//for b
			else if (ix == nx - 1 && iy < ny) {
				d_b[iy] = 0.0f;
			}
		}
		else {//for dense
			if (ix < nx && iy < ny)
				d_W[idx] = 0.0f;
		}
	}


	void BasicLayer::randInit() {
		int m = hiddenStates + 1;
		int n = embedSize + hiddenStates;
		hipMalloc((void**)& rndstates, m * n * sizeof(hiprandState));
		dim3 block(BLOCK_SIZE, 1);
		dim3 grid((n + block.x - 1) / block.x, m);
		randInitGPU << <grid,block >> > (n, m, rndstates);
	}

	void BasicLayer::weightbiasTruncInit(float* Wh, float* Wx, float* b,
		const int Whlen, const int Wxlen, const int blen) {
		randInit();

		float* d_Wh, * d_Wx, * d_b;

		//malloc device memory
		CHECK(hipMalloc((void**)& d_Wh, Whlen * sizeof(float)));
		CHECK(hipMalloc((void**)& d_Wx, Wxlen * sizeof(float)));
		CHECK(hipMalloc((void**)& d_b, blen * sizeof(float)));

		//transfer data from host to device
		CHECK(hipMemcpy(d_Wh, Wh, Whlen * sizeof(float), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_Wx, Wx, Wxlen * sizeof(float), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_b, b, blen * sizeof(float), hipMemcpyHostToDevice));

		//randomlize weights and bias
		int m = hiddenStates + 1;
		int n = blen;

		dim3 block(BLOCK_SIZE, 1);
		dim3 grid((n + block.x - 1) / block.x, m);
		weightbiasTruncInitGPU << <grid, block >> > (hiddenStates, embedSize,
			d_Wh, d_Wx, d_b, rndstates);

		//transfer data from device to host
		CHECK(hipMemcpy(Wh, d_Wh, Whlen * sizeof(float), hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(Wx, d_Wx, Wxlen * sizeof(float), hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(b, d_b, blen * sizeof(float), hipMemcpyDeviceToHost));

		//free device memory
		CHECK(hipFree(d_Wh));
		CHECK(hipFree(d_Wx));
		CHECK(hipFree(d_b));

	}

	void BasicLayer::weightbiasGradInit(float* Wh, float* Wx, float* b,
		const int Whlen, const int Wxlen, const int blen) {

		float* d_Wh, * d_Wx, * d_b;
		std::cout << Whlen << Wxlen << blen << std::endl;
		//malloc device memory
		CHECK(hipMalloc((void**)& d_Wh, Whlen * sizeof(float)));
		CHECK(hipMalloc((void**)& d_Wx, Wxlen * sizeof(float)));
		CHECK(hipMalloc((void**)& d_b, blen * sizeof(float)));

		//Init as 0;
		int m = hiddenStates + 1;
		int n = blen;

		dim3 block(BLOCK_SIZE, 1);
		dim3 grid((n + block.x - 1) / block.x, m);
		weightbiasGradInitGPU << <grid, block >> > (hiddenStates, embedSize,
			d_Wh, d_Wx, d_b);

		//transfer data from device to host
		CHECK(hipMemcpy(Wh, d_Wh, Whlen * sizeof(float), hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(Wx, d_Wx, Wxlen * sizeof(float), hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(b, d_b, blen * sizeof(float), hipMemcpyDeviceToHost));

		//free device memory
		CHECK(hipFree(d_Wh));
		CHECK(hipFree(d_Wx));
		CHECK(hipFree(d_b));

	}

	float* BasicLayer::concatVec(float* vecA, float* vecB,
		const int alen, const int blen) {
		const int outlen = alen + blen;
		float* out = new float[outlen];
		
		float* d_A, * d_B, * d_out;

		//malloc device memory
		CHECK(hipMalloc((void**)& d_A, alen * sizeof(float)));
		CHECK(hipMalloc((void**)& d_B, blen * sizeof(float)));
		CHECK(hipMalloc((void**)& d_out, outlen * sizeof(float)));

		//transfer data from host to device
		CHECK(hipMemcpy(d_A, vecA, alen * sizeof(float), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_B, vecB, blen * sizeof(float), hipMemcpyHostToDevice));

		//Invoke kernel
		dim3 block(BLOCK_SIZE);
		dim3 grid((outlen + block.x - 1) / block.x);
		concatVecGPU << <grid, block >> > (alen, blen, d_out, d_A, d_B);

		//transfer data from device to host
		CHECK(hipMemcpy(out, d_out, outlen * sizeof(float), hipMemcpyDeviceToHost));

		//free device memory
		CHECK(hipFree(d_A));
		CHECK(hipFree(d_B));
		CHECK(hipFree(d_out));

		return out;

	}
	void BasicLayer::showVar() const {
		for (int i = 0; i < 4; i++) {
			std::cout << allVar[i];
		}
		std::cout << std::endl;
	}
	void BasicLayer::showConcat(float* vec, const int len) const {
		for (int i = 0; i < len; i++) {
			std::cout << vec[i] << " ";
		}
		std::cout << std::endl;
	}

	void BasicLayer::denseweightbiasTruncInit(float* W, float* b,
		const int Wlen, const int blen) {
		randInit();

		float* d_W, * d_b;

		//malloc device memory
		CHECK(hipMalloc((void**)& d_W, Wlen * sizeof(float)));
		CHECK(hipMalloc((void**)& d_b, blen * sizeof(float)));

		//transfer data from host to device
		CHECK(hipMemcpy(d_W, W, Wlen * sizeof(float), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_b, b, blen * sizeof(float), hipMemcpyHostToDevice));

		//randomlize weights and bias
		int m = categories;
		int n = hiddenStates + 1;

		dim3 block(BLOCK_SIZE, 1);
		dim3 grid((n + block.x - 1) / block.x, m);
		denseweightbiasTruncInitGPU << <grid, block >> > (n, m,
			 d_W, d_b, rndstates, 1);

		//transfer data from device to host
		CHECK(hipMemcpy(W, d_W, Wlen * sizeof(float), hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(b, d_b, blen * sizeof(float), hipMemcpyDeviceToHost));

		//free device memory
		CHECK(hipFree(d_W));
		CHECK(hipFree(d_b));

	}

	void BasicLayer::denseweightbiasGradInit(float* W, float* b,
		const int Wlen, const int blen) {
		randInit();

		float* d_W, * d_b;

		//malloc device memory
		CHECK(hipMalloc((void**)& d_W, Wlen * sizeof(float)));
		CHECK(hipMalloc((void**)& d_b, blen * sizeof(float)));

		//transfer data from host to device
		CHECK(hipMemcpy(d_W, W, Wlen * sizeof(float), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_b, b, blen * sizeof(float), hipMemcpyHostToDevice));

		//randomlize weights and bias
		int m = categories;
		int n = hiddenStates + 1;

		dim3 block(BLOCK_SIZE, 1);
		dim3 grid((n + block.x - 1) / block.x, m);
		denseweightbiasGradInitGPU << <grid, block >> > (n, m,
			d_W, d_b, rndstates, 1);

		//transfer data from device to host
		CHECK(hipMemcpy(W, d_W, Wlen * sizeof(float), hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(b, d_b, blen * sizeof(float), hipMemcpyDeviceToHost));

		//free device memory
		CHECK(hipFree(d_W));
		CHECK(hipFree(d_b));
	}

	void BasicLayer::embedweightTruncInit(float* W, const int Wlen) {

		randInit();

		float* d_W;

		//malloc device memory
		CHECK(hipMalloc((void**)& d_W, Wlen * sizeof(float)));

		//transfer data from host to device
		CHECK(hipMemcpy(d_W, W, Wlen * sizeof(float), hipMemcpyHostToDevice));

		//randomlize weights and bias
		int m = embedSize;
		int n = Wlen / m;

		dim3 block(BLOCK_SIZE, 1);
		dim3 grid((n + block.x - 1) / block.x, m);
		denseweightbiasTruncInitGPU << <grid, block >> > (n, m,
			d_W, d_W, rndstates, 0);

		//transfer data from device to host
		CHECK(hipMemcpy(W, d_W, Wlen * sizeof(float), hipMemcpyDeviceToHost));

		//free device memory
		CHECK(hipFree(d_W));

	}

	void BasicLayer::embedweightGradInit(float* W, const int Wlen) {

		randInit();

		float* d_W;

		//malloc device memory
		CHECK(hipMalloc((void**)& d_W, Wlen * sizeof(float)));

		//transfer data from host to device
		CHECK(hipMemcpy(d_W, W, Wlen * sizeof(float), hipMemcpyHostToDevice));

		//randomlize weights and bias
		int m = embedSize;
		int n = Wlen / m;

		dim3 block(BLOCK_SIZE, 1);
		dim3 grid((n + block.x - 1) / block.x, m);
		denseweightbiasGradInitGPU << <grid, block >> > (n, m,
			d_W, d_W, rndstates, 0);

		//transfer data from device to host
		CHECK(hipMemcpy(W, d_W, Wlen * sizeof(float), hipMemcpyDeviceToHost));

		//free device memory
		CHECK(hipFree(d_W));
	}

}