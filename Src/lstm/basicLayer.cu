#include "hip/hip_runtime.h"
/**
*	Author: Lingfeng Chen, Pengkun Li
*   PID: A53270085, A53270024
*	This file defines the data structure for grid
*/
#include "basicLayer.h"
#define BLOCK_SIZE 32

namespace basicLayer {

	__global__ void randInitGPU(int nx, int ny, hiprandState* rndstates) {
		unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
		unsigned int iy = blockIdx.y;
		unsigned int idx = iy * nx + ix;
		if (ix < nx && iy < ny) {
			hiprand_init(clock() + idx, idx, 0, &rndstates[idx]);
		}
			
	}

	__global__ void weightbiasTruncInitGPU(int hid, int emb, float* d_Wh, float* d_Wx,
		float* d_b, hiprandState* rndstates) { // truncated uniform
		unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
		unsigned int iy = blockIdx.y;
		unsigned int idh = iy * (hid + emb) + ix;

		//for Wh
		if (ix == hid + emb - 1 && iy == hid)
			printf("%d, %d\n", iy, ix);
		if (ix < hid && iy < hid) {
			d_Wh[iy * hid + ix] = hiprand_uniform(&rndstates[idh]) * 2.0E-4 - 1.0E-4;
		}//for Wx
		else if (ix < hid + emb && iy < hid) {
			d_Wx[iy * emb + ix - hid] = hiprand_uniform(&rndstates[idh]) * 2.0E-4 - 1.0E-4;
		}//for bin
		else if (ix < hid && iy == hid) {
			d_b[ix] = hiprand_uniform(&rndstates[idh]) * 2.0E-4 - 1.0E-4;
		}

	}
	__global__ void concatVecGPU(const int nx, const int ny, float* d_out, 
		float* d_A, float* d_B){
		unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx < nx) {
			d_out[idx] = d_A[idx];
		}
		else if (idx < nx + ny) {
			d_out[idx] = d_B[idx - nx];
		}
	
	}

	void BasicLayer::randInit() {
		int m = hiddenStates + 1;
		int n = embedSize + hiddenStates;
		hipMalloc((void**)& rndstates, m * n * sizeof(hiprandState));
		dim3 block(BLOCK_SIZE, 1);
		dim3 grid((n + block.x - 1) / block.x, m);
		randInitGPU << <grid,block >> > (n, m, rndstates);
	}

	void BasicLayer::weightbiasTruncInit(float* Wh, float* Wx, float* b,
		const int Whlen, const int Wxlen, const int blen) {
		randInit();

		float* d_Wh, * d_Wx, * d_b;
		std::cout << Whlen << Wxlen << blen<<std::endl;
		//malloc device memory
		CHECK(hipMalloc((void**)& d_Wh, Whlen * sizeof(float)));
		CHECK(hipMalloc((void**)& d_Wx, Wxlen * sizeof(float)));
		CHECK(hipMalloc((void**)& d_b, blen * sizeof(float)));

		//transfer data from host to device
		CHECK(hipMemcpy(d_Wh, Wh, Whlen * sizeof(float), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_Wx, Wx, Wxlen * sizeof(float), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_b, b, blen * sizeof(float), hipMemcpyHostToDevice));

		//randomlize weights and bias
		int m = hiddenStates + 1;
		int n = blen;

		dim3 block(BLOCK_SIZE, 1);
		dim3 grid((n + block.x - 1) / block.x, m);
		weightbiasTruncInitGPU << <grid, block >> > (hiddenStates, embedSize,
			d_Wh, d_Wx, d_b, rndstates);

		//transfer data from device to host
		CHECK(hipMemcpy(Wh, d_Wh, Whlen * sizeof(float), hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(Wx, d_Wx, Wxlen * sizeof(float), hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(b, d_b, blen * sizeof(float), hipMemcpyDeviceToHost));

		//free device memory
		CHECK(hipFree(d_Wh));
		CHECK(hipFree(d_Wx));
		CHECK(hipFree(d_b));

	}

	float* BasicLayer::concatVec(float* vecA, float* vecB,
		const int alen, const int blen) {
		const int outlen = alen + blen;
		float* out = new float[outlen];
		
		float* d_A, * d_B, * d_out;

		//malloc device memory
		CHECK(hipMalloc((void**)& d_A, alen * sizeof(float)));
		CHECK(hipMalloc((void**)& d_B, blen * sizeof(float)));
		CHECK(hipMalloc((void**)& d_out, outlen * sizeof(float)));

		//transfer data from host to device
		CHECK(hipMemcpy(d_A, vecA, alen * sizeof(float), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_B, vecB, blen * sizeof(float), hipMemcpyHostToDevice));

		//Invoke kernel
		dim3 block(BLOCK_SIZE);
		dim3 grid((outlen + block.x - 1) / block.x);
		concatVecGPU << <grid, block >> > (alen, blen, d_out, d_A, d_B);

		//transfer data from device to host
		CHECK(hipMemcpy(out, d_out, outlen * sizeof(float), hipMemcpyDeviceToHost));

		//free device memory
		CHECK(hipFree(d_A));
		CHECK(hipFree(d_B));
		CHECK(hipFree(d_out));

		return out;

	}
	void BasicLayer::showVar() const {
		for (int i = 0; i < 4; i++) {
			std::cout << allVar[i];
		}
		std::cout << std::endl;
	}
	void BasicLayer::showConcat(float* vec, const int len) const {
		for (int i = 0; i < len; i++) {
			std::cout << vec[i] << " ";
		}
		std::cout << std::endl;
	}
}