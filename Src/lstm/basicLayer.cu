#include "hip/hip_runtime.h"
/**
*	Author: Lingfeng Chen, Pengkun Li
*   PID: A53270085, A53270024
*	This file defines the data structure for grid
*/
#include "basicLayer.h"
#define BLOCK_SIZE 32

namespace basicLayer {

	__global__ void randInitGPU(int nx, int ny, hiprandState* rndstates) {
		unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
		unsigned int iy = blockIdx.y;
		unsigned int idx = iy * nx + ix;
		if (ix < nx && iy < ny) {
			hiprand_init(clock() + idx, idx, 0, &rndstates[idx]);
		}
			
	}

	__global__ void weightbiasTruncInitGPU(int nx, int ny, float* d_W,
		float* d_b, hiprandState* rndstates) { // truncated uniform
		unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
		unsigned int iy = blockIdx.y;
		unsigned int idx = iy * nx + ix;

		//for Win
		if (ix < nx && iy < ny - 1) {
			d_W[idx] = hiprand_uniform(&rndstates[idx]) * 2.0E-4 - 1.0E-4;
		}//for bin
		else if (ix < nx && iy == ny - 1) {
			d_b[ix] = hiprand_uniform(&rndstates[idx]) * 2.0E-4 - 1.0E-4;
		}

	}
	__global__ void concatVecGPU(const int nx, const int ny, float* d_out, 
		float* d_A, float* d_B){
		unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx < nx) {
			d_out[idx] = d_A[idx];
		}
		else if (idx < nx + ny) {
			d_out[idx] = d_B[idx - nx];
		}
	
	}

	void BasicLayer::randInit() {
		int m = hiddenStates + 1;
		int n = embedSize + hiddenStates;
		hipMalloc((void**)& rndstates, m * n * sizeof(hiprandState));
		dim3 block(BLOCK_SIZE, 1);
		dim3 grid((n + block.x - 1) / block.x, m);
		randInitGPU << <grid,block >> > (n, m, rndstates);
	}

	void BasicLayer::weightbiasTruncInit(float* W, float* b, const int Wlen, const int blen) {
		randInit();

		float* d_W, * d_b;

		//malloc device memory
		CHECK(hipMalloc((void**)& d_W, Wlen * sizeof(float)));
		CHECK(hipMalloc((void**)& d_b, blen * sizeof(float)));

		//transfer data from host to device
		CHECK(hipMemcpy(d_W, W, Wlen * sizeof(float), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_b, b, blen * sizeof(float), hipMemcpyHostToDevice));

		//randomlize weights and bias
		int m = hiddenStates + 1;
		int n = blen;

		dim3 block(BLOCK_SIZE, 1);
		dim3 grid((n + block.x - 1) / block.x, m);
		weightbiasTruncInitGPU << <grid, block >> > (n, m, d_W, d_b, rndstates);

		//transfer data from device to host
		CHECK(hipMemcpy(W, d_W, Wlen * sizeof(float), hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(b, d_b, blen * sizeof(float), hipMemcpyDeviceToHost));

		//free device memory
		CHECK(hipFree(d_W));
		CHECK(hipFree(d_b));

	}

	float* BasicLayer::concatVec(float* vecA, float* vecB,
		const int alen, const int blen) {
		const int outlen = alen + blen;
		float* out = new float[outlen];
		
		float* d_A, * d_B, * d_out;

		//malloc device memory
		CHECK(hipMalloc((void**)& d_A, alen * sizeof(float)));
		CHECK(hipMalloc((void**)& d_B, blen * sizeof(float)));
		CHECK(hipMalloc((void**)& d_out, outlen * sizeof(float)));

		//transfer data from host to device
		CHECK(hipMemcpy(d_A, vecA, alen * sizeof(float), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_B, vecB, blen * sizeof(float), hipMemcpyHostToDevice));

		//Invoke kernel
		dim3 block(BLOCK_SIZE);
		dim3 grid((outlen + block.x - 1) / block.x);
		concatVecGPU << <grid, block >> > (alen, blen, d_out, d_A, d_B);

		//transfer data from device to host
		CHECK(hipMemcpy(out, d_out, outlen * sizeof(float), hipMemcpyDeviceToHost));

		//free device memory
		CHECK(hipFree(d_A));
		CHECK(hipFree(d_B));
		CHECK(hipFree(d_out));

		return out;

	}
	void BasicLayer::showVar() const {
		for (int i = 0; i < 4; i++) {
			std::cout << allVar[i];
		}
		std::cout << std::endl;
	}
	void BasicLayer::showConcat(float* vec, const int len) const {
		for (int i = 0; i < len; i++) {
			std::cout << vec[i] << " ";
		}
		std::cout << std::endl;
	}
}