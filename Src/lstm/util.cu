#include "hip/hip_runtime.h"
/**
*	Author: Lingfeng Chen, Pengkun Li
*   PID: A53270085, A53270024
*	This file defines the data structure for grid
*/
#include <vector>
#include <iostream>

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "util.h"

namespace util {
	__global__ void cudaHello()
	{
		printf("Hello World from GPU!\n");
	}

	void hellofromGPU()
	{
		cudaHello << <1, 10 >> > ();
	}
}