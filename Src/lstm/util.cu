#include "hip/hip_runtime.h"
/**
*	Author: Lingfeng Chen, Pengkun Li
*   PID: A53270085, A53270024
*	This file defines the data structure for grid
*/
#include <vector>
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "util.h"
#include "../common/common.h"
#define BLOCK_SIZE 32
#define CATEGORIES 3


namespace util {

	__global__ void matrixSumGPU(float* d_A, float* d_B, int nx, int ny) {
		unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
		unsigned int iy = blockIdx.y;
		unsigned int idx = iy * nx + ix;
		if (ix < nx && iy < ny)
			d_A[idx] += d_B[idx];
	}

	__global__ void matrixMulElemGPU(float* d_A, float* d_B, int nx, int ny) {
		unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
		unsigned int iy = blockIdx.y;
		unsigned int idx = iy * nx + ix;
		if (ix < nx && iy < ny)
			d_A[idx] *= d_B[idx];
	}

	__global__ void matrixMulGPU(float* out, float* d_A, float* d_B, int ny, int nx, int nz)
	{
		int iy = blockIdx.y * blockDim.y + threadIdx.y;
		int iz = blockIdx.x * blockDim.x + threadIdx.x;
		float sum = 0;
		if (iz < nz && iy < ny)
		{
			for (int ix = 0; ix < nx; ix++)
			{
				sum += d_A[iy * nx + ix] * d_B[ix * nz + iz];
			}
			out[iy * nz + iz] = sum;
		}
	}

	__global__ void softmaxGPU(float *d_A, int m) 
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx < m) {
			float exp = expf(d_A[idx]);
			d_A[idx] = exp;
			float sum = 0;


			// compute sum
			int startIdx = idx / CATEGORIES;
			int end = startIdx + CATEGORIES;
			for (int i = startIdx; i < end; ++i) {
				sum += d_A[i];
			}

			float sm = exp / sum;

			d_A[idx] = sm;
		}
	}

	__global__ void tanhActivation(float* d_A, int m)
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		float exp1 = expf(d_A[idx]);
		float exp2 = expf(-d_A[idx]);
		if (idx < m) {
			d_A[idx] = (exp1 - exp2) / (exp1 + exp2);
		}
	}

	__global__ void sigmoidActivation(float *d_A, int m)
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		float exp = expf(d_A[idx]);
		if(idx < m){
			d_A[idx] = exp / (1.0f + exp);
		}
		
	}




	void matrixSum(float* matA, float* matB, int m, int n) {
		float* d_A, *d_B;

		//malloc device memory
		CHECK(hipMalloc((void**)& d_A, m * n * sizeof(float)));
		CHECK(hipMalloc((void**)& d_B, m * n * sizeof(float)));

		//transfer data from host to device
		CHECK(hipMemcpy(d_A, matA, m * n * sizeof(float), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_B, matB, m * n * sizeof(float), hipMemcpyHostToDevice));

		// invoke kernel
		dim3 block(BLOCK_SIZE,1);
		dim3 grid((n + block.x - 1) / block.x, m);
		matrixSumGPU << <grid, block >> > (d_A, d_B, n, m);

		//transfer data from device to host
		CHECK(hipMemcpy(matA, d_A, m * n * sizeof(float), hipMemcpyDeviceToHost));

		//free device memory
		CHECK(hipFree(d_A));
		CHECK(hipFree(d_B));
	}

	void matrixMulElem(float* matA, float* matB, int m, int n) {
		float* d_A, * d_B;

		//malloc device memory
		CHECK(hipMalloc((void**)& d_A, m * n * sizeof(float)));
		CHECK(hipMalloc((void**)& d_B, m * n * sizeof(float)));

		//transfer data from host to device
		CHECK(hipMemcpy(d_A, matA, m * n * sizeof(float), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_B, matB, m * n * sizeof(float), hipMemcpyHostToDevice));

		// invoke kernel
		dim3 block(BLOCK_SIZE, 1);
		dim3 grid((n + block.x - 1) / block.x, m);
		matrixMulElemGPU << <grid, block >> > (d_A, d_B, n, m);

		//transfer data from device to host
		CHECK(hipMemcpy(matA, d_A, m * n * sizeof(float), hipMemcpyDeviceToHost));

		//free device memory
		CHECK(hipFree(d_A));
		CHECK(hipFree(d_B));
	}

	void matrixMul(float* out, float* matA, float* matB, int m, int n, int k) {
		float* d_out, * d_A, * d_B;
		CHECK(hipMalloc((void**)& d_out, m * k * sizeof(float)));
		CHECK(hipMalloc((void**)& d_A, m * n * sizeof(float)));
		CHECK(hipMalloc((void**)& d_B, n * k * sizeof(float)));

		CHECK(hipMemcpy(d_A, matA, m * n * sizeof(float), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_B, matB, n * k * sizeof(float), hipMemcpyHostToDevice));

		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid((k + block.x - 1) / block.x, (m + block.y - 1) / block.y);

		matrixMulGPU << <grid, block >> > (d_out, d_A, d_B, m, n, k);
		CHECK(hipMemcpy(out, d_out, m * k * sizeof(float), hipMemcpyDeviceToHost));
		hipFree(d_out);
		hipFree(d_A);
		hipFree(d_B);
	}

	void softmax(float* A, int len){
		float *d_A;
		CHECK(hipMalloc((void**)&d_A, len * sizeof(float)));
		CHECK(hipMemcpy(d_A, A, len * sizeof(float), hipMemcpyHostToDevice));
		dim3 block(BLOCK_SIZE);
		dim3 grid((len + block.x - 1) / block.x);
		softmaxGPU << <grid, block >> > (d_A, len);
		CHECK(hipMemcpy(A, d_A, len * sizeof(float), hipMemcpyDeviceToHost));
		hipFree(d_A);
	
	}
	
	void tanh(float* A, int len){

		float *d_A;
		CHECK(hipMalloc((void**)&d_A, len * sizeof(float)));
		CHECK(hipMemcpy(d_A, A, len * sizeof(float), hipMemcpyHostToDevice));
		dim3 block(BLOCK_SIZE);
		dim3 grid((len + block.x - 1) / block.x);
		tanhActivation << <grid, block >> > (d_A, len);
		CHECK(hipMemcpy(A, d_A, len * sizeof(float), hipMemcpyDeviceToHost));
		hipFree(d_A);
	
	}
	
	
	void sigmoid(float *A, int len){
		
		float *d_A;
		CHECK(hipMalloc((void**)&d_A, len * sizeof(float)));
		CHECK(hipMemcpy(d_A, A, len * sizeof(float), hipMemcpyHostToDevice));
		dim3 block(BLOCK_SIZE);
		dim3 grid((len + block.x - 1) / block.x);
		sigmoidActivation << <grid, block >> > (d_A, len);
		CHECK(hipMemcpy(A, d_A, len * sizeof(float), hipMemcpyDeviceToHost));
		hipFree(d_A);
	}
}