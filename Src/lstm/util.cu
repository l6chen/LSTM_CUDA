#include "hip/hip_runtime.h"
/**
*	Author: Lingfeng Chen, Pengkun Li
*   PID: A53270085, A53270024
*	This file defines the data structure for grid
*/
#include <vector>
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "util.h"
#include "../common/common.h"
#define BLOCK_SIZE 32

namespace util {

	__global__ void matrixSumGPU(float* d_A, float* d_B, int nx, int ny) {
		unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
		unsigned int iy = blockIdx.y;
		unsigned int idx = iy * nx + ix;
		if (ix < nx && iy < ny)
			d_A[idx] += d_B[idx];
	}

	__global__ void matrixMulElemGPU(float* d_A, float* d_B, int nx, int ny) {
		unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
		unsigned int iy = blockIdx.y;
		unsigned int idx = iy * nx + ix;
		if (ix < nx && iy < ny)
			d_A[idx] *= d_B[idx];
	}

	__global__ void matrixMulGPU(float* out, float* d_A, float* d_B, int ny, int nx, int nz)
	{
		int iy = blockIdx.y * blockDim.y + threadIdx.y;
		int iz = blockIdx.x * blockDim.x + threadIdx.x;
		float sum = 0;
		if (iz < nz && iy < ny)
		{
			for (int ix = 0; ix < nx; ix++)
			{
				sum += d_A[iy * nx + ix] * d_B[ix * nz + iz];
			}
			out[iy * nz + iz] = sum;
		}
	}

	void matrixSum(float* matA, float* matB, int m, int n) {
		float* d_A, *d_B;

		//malloc device memory
		CHECK(hipMalloc((void**)& d_A, m * n * sizeof(float)));
		CHECK(hipMalloc((void**)& d_B, m * n * sizeof(float)));

		//transfer data from host to device
		CHECK(hipMemcpy(d_A, matA, m * n * sizeof(float), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_B, matB, m * n * sizeof(float), hipMemcpyHostToDevice));

		// invoke kernel
		dim3 block(BLOCK_SIZE,1);
		dim3 grid((n + block.x - 1) / block.x, m);
		matrixSumGPU << <grid, block >> > (d_A, d_B, n, m);

		//transfer data from device to host
		CHECK(hipMemcpy(matA, d_A, m * n * sizeof(float), hipMemcpyDeviceToHost));

		//free device memory
		CHECK(hipFree(d_A));
		CHECK(hipFree(d_B));
	}

	void matrixMulElem(float* matA, float* matB, int m, int n) {
		float* d_A, * d_B;

		//malloc device memory
		CHECK(hipMalloc((void**)& d_A, m * n * sizeof(float)));
		CHECK(hipMalloc((void**)& d_B, m * n * sizeof(float)));

		//transfer data from host to device
		CHECK(hipMemcpy(d_A, matA, m * n * sizeof(float), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_B, matB, m * n * sizeof(float), hipMemcpyHostToDevice));

		// invoke kernel
		dim3 block(BLOCK_SIZE, 1);
		dim3 grid((n + block.x - 1) / block.x, m);
		matrixMulElemGPU << <grid, block >> > (d_A, d_B, n, m);

		//transfer data from device to host
		CHECK(hipMemcpy(matA, d_A, m * n * sizeof(float), hipMemcpyDeviceToHost));

		//free device memory
		CHECK(hipFree(d_A));
		CHECK(hipFree(d_B));
	}

	void matrixMul(float* out, float* matA, float* matB, int m, int n, int k) {
		float* d_out, * d_A, * d_B;
		CHECK(hipMalloc((void**)& d_out, m * k * sizeof(float)));
		CHECK(hipMalloc((void**)& d_A, m * n * sizeof(float)));
		CHECK(hipMalloc((void**)& d_B, n * k * sizeof(float)));

		CHECK(hipMemcpy(d_A, matA, m * n * sizeof(float), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_B, matB, n * k * sizeof(float), hipMemcpyHostToDevice));

		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid((k + block.x - 1) / block.x, (m + block.y - 1) / block.y);

		matrixMulGPU << <grid, block >> > (d_out, d_A, d_B, m, n, k);
		CHECK(hipMemcpy(out, d_out, m * k * sizeof(float), hipMemcpyDeviceToHost));
		hipFree(d_out);
		hipFree(d_A);
		hipFree(d_B);
	}

	void softmax(){}
	void tanh(){}
	void sigmoid(){}
}