#include "hip/hip_runtime.h"
/**
*	Author: Lingfeng Chen, Pengkun Li
*   PID: A53270085, A53270024
*	This file defines the data structure for grid
*/
#include "test_util.h"
#include <string>
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

namespace testUtil {
	void initialData(float* ip, const int size)
	{

		for (int i = 0; i < size; i++)
		{
			ip[i] = (float)(rand() & 0xFF) / 10.0f;
		}

		return;
	}

	void sumMatrixOnHost(float* A, float* B, float* C, const int nx,
		const int ny)
	{
		float* ia = A;
		float* ib = B;
		float* ic = C;

		for (int iy = 0; iy < ny; iy++)
		{
			for (int ix = 0; ix < nx; ix++)
			{
				ic[ix] = ia[ix] + ib[ix];

			}

			ia += nx;
			ib += nx;
			ic += nx;
		}

		return;
	}

	void mulElemMatrixOnHost(float* A, float* B, float* C, const int nx,
		const int ny)
	{
		float* ia = A;
		float* ib = B;
		float* ic = C;

		for (int iy = 0; iy < ny; iy++)
		{
			for (int ix = 0; ix < nx; ix++)
			{
				ic[ix] = ia[ix] * ib[ix];

			}

			ia += nx;
			ib += nx;
			ic += nx;
		}

		return;
	}

	void mulMatrixOnHost(float* A, float* B, float* C, const int nx,
		const int ny, const int nz)
	{
		float* ia = A;
		float* ib = B;
		float* ic = C;

		for (int iy = 0; iy < ny; iy++)
		{
			for (int iz = 0; iz < nz; iz++)
			{
				float sum = 0;
				for (int ix = 0; ix < nx; ix++)
					sum += ia[iy * nx + ix] * ib[ix * nz + iz];
				ic[iy * nz + iz] = sum;
			}
		}

		return;
	}

	void checkResult(float* hostRef, float* gpuRef, const int N, std::string testtype)
	{
		double epsilon = 1.0E-8;
		bool match = 1;

		for (int i = 0; i < N; i++)
		{
			if (abs(hostRef[i] - gpuRef[i]) > epsilon)
			{
				match = 0;
				printf("host %f gpu %f\n", hostRef[i], gpuRef[i]);
				break;
			}
		}

		if (match)
			std::cout << "Arrays match for " << testtype << "\n\n";
		else
			std::cout << "Arrays do not match for " << testtype << "\n\n";
	}

	void testmatrixSum() {

		std::string testtype = "Sum";
		int nx = 1 << 14;
		int ny = 1 << 14;

		int nxy = nx * ny;
		int nBytes = nxy * sizeof(float);
		
		float* matA, * matB, * matC;
		matA = (float*)malloc(nBytes);
		matB = (float*)malloc(nBytes);
		matC = (float*)malloc(nBytes);

		initialData(matA, nxy);
		initialData(matB, nxy);

		sumMatrixOnHost(matA, matB, matC, nx, ny);
		util::matrixSum(matA, matB, ny, nx);
		checkResult(matC, matA, nxy, testtype);

		// free host memory
		free(matA);
		free(matB);
		free(matC);

		// reset device
		CHECK(hipDeviceReset());
		
	}

	void testmatrixMulElem() {

		std::string testtype = "MulElem";
		int nx = 1 << 14;
		int ny = 1 << 14;

		int nxy = nx * ny;
		int nBytes = nxy * sizeof(float);

		float* matA, * matB, * matC;
		matA = (float*)malloc(nBytes);
		matB = (float*)malloc(nBytes);
		matC = (float*)malloc(nBytes);

		initialData(matA, nxy);
		initialData(matB, nxy);

		mulElemMatrixOnHost(matA, matB, matC, nx, ny);
		util::matrixMulElem(matA, matB, ny, nx);
		checkResult(matC, matA, nxy, testtype);

		// free host memory
		free(matA);
		free(matB);
		free(matC);

		// reset device
		CHECK(hipDeviceReset());

	}

	void testmatrixMul() {

		std::string testtype = "Mul";
		int ny = 1 << 3;
		int nx = 1 << 4;
		int nz = 1 << 5;

		int nxy = nx * ny, nyz = ny * nz, nxz = nx * nz;
		int nxyB = nxy * sizeof(float), nyzB = nyz * sizeof(float),
			nxzB = nxz * sizeof(float);

		float* matA, * matB, * cpuM, *gpuM;
		matA = (float*)malloc(nxyB);
		matB = (float*)malloc(nxzB);
		cpuM = (float*)malloc(nyzB);
		gpuM = (float*)malloc(nyzB);

		initialData(matA, nxy);
		initialData(matB, nxz);

		mulMatrixOnHost(matA, matB, cpuM, nx, ny, nz);
		util::matrixMul(gpuM, matA, matB, ny, nx, nz);
		checkResult(cpuM, gpuM, nyz, testtype);

		// free host memory
		free(matA);
		free(matB);
		free(cpuM);
		free(gpuM);

		// reset device
		CHECK(hipDeviceReset());

	}
}
