#include "hip/hip_runtime.h"
/**
*	Author: Lingfeng Chen, Pengkun Li
*   PID: A53270085, A53270024
*	This file defines the data structure for grid
*/
#include <iostream>
#include <string>
#include <vector>
#include "../lstm/dataLoader.h"
#include "test_util.h"
#include "test_Layer.h"




int main() {
	const std::string DELIMITER = "================================================================================";
	std::cout << "Test begins." << std::endl;

	//Test Dataloader
	std::cout << DELIMITER << std::endl << "Test Data Loader" << std::endl;

	dataLoader::DataLoader loader(0.8);
	dataLoader::DataSets* ds = loader.load();

	//Test Elementwise operations
	std::cout << DELIMITER << std::endl << "Test Elementwise operations" << std::endl;
	testUtil::testmatElem('+');
	testUtil::testmatElem('-');
	testUtil::testmatElem('*');
	//testUtil::testmatElem('/');//not supported

	//Test Matrix Mul
	std::cout << DELIMITER << std::endl << "Test Matrix Mul" << std::endl;
	testUtil::testmatMul();

	//Test Matrix Scal Mul
	std::cout << DELIMITER << std::endl << "Test Matrix Scal Mul" << std::endl;
	testUtil::testmatMulScal();
	
	//Test transpose
	std::cout << DELIMITER << std::endl << "Test Matrix Tanspose" << std::endl;
	testUtil::testmatTranspose();	

	//Test tanh
	std::cout << DELIMITER << std::endl << "Test tanh" << std::endl;
	testUtil::testtanh();

	//Test softmax
	std::cout << DELIMITER << std::endl << "Test softmax" << std::endl;
	testUtil::testsoftmax();

	//test sigmoid
	std::cout << DELIMITER << std::endl << "Test sigmoid" << std::endl;
	testUtil::testsigmoid();

	//Test Tanh Prime
	std::cout << DELIMITER << std::endl << "Test Tanh Prime" << std::endl;
	testUtil::testtanhPrime();

	//Test Sigmoid Prime
	std::cout << DELIMITER << std::endl << "Test Sigmoid Prime" << std::endl;
	testUtil::testsigmoidPrime();

	//Test Cross Entropy Loss
	std::cout << DELIMITER << std::endl << "Test Cross Entropy Loss" << std::endl;
	testUtil::testcrossEntropyLoss();

	//Test Basic Layer
	std::cout << DELIMITER << std::endl << "Test Basic Layer" << std::endl;
	testLayer::testBasicLayer();

	//Test Gate Layer
	std::cout << DELIMITER << std::endl << "Test Gate Layer" << std::endl;
	testLayer::testGateLayer();

	//Test Dense Layer
	std::cout << DELIMITER << std::endl << "Test Dense Layer" << std::endl;
	testLayer::testDenseLayer();
	
	//Test Embed Layer
	std::cout << DELIMITER << std::endl << "Test Embed Layer" << std::endl;
	testLayer::testEmbedLayer();

	std::cout << "Test ends.";
	system("PAUSE");
	return 0;
}